#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>

#include "csr.hpp"

__global__ void bf_iteration(int n, int s,
                             unsigned int *csr_index, unsigned int *csr_cols, float *csr_weights,
                             float *d, float *d_new, int *result) {
    auto thisThread = blockIdx.x * blockDim.x + threadIdx.x;
    auto numThreads = gridDim.x + blockDim.x;
    auto indexAddition = n*s;
    bool changes = false;
    for (unsigned int v = thisThread; v < n; v += numThreads) {
        float dist = d[v+indexAddition];
        for(unsigned int i = csr_index[v]; i < csr_index[v + 1]; ++i) {
            auto u = csr_cols[i];
            auto weight = csr_weights[i];

            if(dist > d[u+indexAddition] + weight) {
                dist = d[u+indexAddition] + weight;
                changes = true;
            }
        }
        d_new[v+indexAddition] = dist;
    }
    if(changes)
        *result = 1;
}


void run_bf(const csr_matrix &tr, unsigned int batchsize,
		const std::vector<unsigned int> &sources) {
	// TODO
    unsigned int num_blocks = (tr.n + 255) / 256;
    unsigned int n_sources = sources.size();
    unsigned int *csr_index;
    unsigned int *csr_cols;
    float *csr_weights;
    float *d;
    float *d_new;
    int *result;

    hipMalloc(&csr_index, (tr.n + 1) * sizeof(unsigned int));
    hipMalloc(&csr_cols, tr.nnz * sizeof(unsigned int));
    hipMalloc(&csr_weights, tr.nnz * sizeof(float));
    hipMalloc(&d, tr.n * n_sources * sizeof(float));
    hipMalloc(&d_new, tr.n * n_sources * sizeof(float));
    hipMalloc(&result, sizeof(int));

    hipMemcpy(csr_index, tr.ind.data(), (tr.n + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csr_cols, tr.cols.data(), tr.nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csr_weights, tr.weights.data(), tr.nnz * sizeof(unsigned int), hipMemcpyHostToDevice);

    auto algo_start = std::chrono::high_resolution_clock::now();
    std::vector<float> initial;
    initial.resize(tr.n*n_sources);
    std::fill(initial.begin(), initial.end(), FLT_MAX);

    for(unsigned int i = 0; i < sources.size(); ++i) {
        initial[sources[i] + i * tr.n] = 0;
    }
    hipMemcpy(d, initial.data(), n_sources * tr.n * sizeof(float), hipMemcpyHostToDevice);

    for(unsigned int i = 0; i< sources.size(); ++i){
        while(true) {
            hipMemset(result, 0, sizeof(int));
            bf_iteration<<<num_blocks, 256>>>(tr.n, i, csr_index, csr_cols, csr_weights,
                                              d, d_new, result);

            unsigned int c;
            hipMemcpy(&c, result, sizeof(int), hipMemcpyDeviceToHost);
            if(!c)
                break;
            std::swap(d, d_new);
        }
    }
    auto t_algo = std::chrono::high_resolution_clock::now() - algo_start;

    std::cout << "time_sssp: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t_algo).count() << std::endl;

    hipFree(csr_index);
    hipFree(csr_cols);
    hipFree(csr_weights);
    hipFree(d);
    hipFree(d_new);
    hipFree(result);

}

int main(int argc, char **argv) {
	if(argc != 3)
		throw std::runtime_error("Expected instance and batch size as argument");

	unsigned int batchsize = std::atoi(argv[2]);

	std::mt19937 prng{42};
	std::uniform_real_distribution<float> weight_distrib{0.0f, 1.0f};

	// Load the graph.
	std::cout << "instance: " << argv[1] << std::endl;
	std::cout << "batchsize: " << batchsize << std::endl;

	std::ifstream ins(argv[1]);
	std::vector<std::tuple<unsigned int, unsigned int, float>> cv;

	auto io_start = std::chrono::high_resolution_clock::now();
	read_graph_unweighted(ins, [&] (unsigned int u, unsigned int v) {
		// Generate a random edge weight in [a, b).
		cv.push_back({u, v, weight_distrib(prng)});
	});

	auto mat = coordinates_to_csr(std::move(cv));
	auto t_io = std::chrono::high_resolution_clock::now() - io_start;

	std::cout << "time_io: "
			<< std::chrono::duration_cast<std::chrono::milliseconds>(t_io).count() << std::endl;
	std::cout << "n_nodes: " << mat.n << std::endl;
	std::cout << "n_edges: " << mat.nnz << std::endl;

	auto tr = transpose(std::move(mat));

	// Generate random sources.
	std::uniform_int_distribution<unsigned int> s_distrib{0, mat.n - 1};
	std::vector<unsigned int> sources;
	for(unsigned int i = 0; i < batchsize; ++i)
		sources.push_back(s_distrib(prng));

	// Run the algorithm.
	auto algo_start = std::chrono::high_resolution_clock::now();
	run_bf(tr, batchsize, sources);
	auto t_algo = std::chrono::high_resolution_clock::now() - algo_start;

	std::cout << "time_mssp: "
			<< std::chrono::duration_cast<std::chrono::milliseconds>(t_algo).count() << std::endl;
}
